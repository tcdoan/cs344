

#include <hip/hip_runtime.h>
#include <getopt.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <cassert>
#include <chrono>

const unsigned int BLOCKS = 6;
const unsigned int BLOCK_DIM = 8;

__global__ void add(float *Y, float *sums, int unsigned n) {
    extern __shared__ float temp[];

    int tid = threadIdx.x;
    int id = blockDim.x * blockIdx.x + tid;

    temp[2 * tid] = Y[2 * id];
    temp[2 * tid + 1] = Y[2 * id + 1];

    temp[2 * tid] += sums[blockIdx.x];
    temp[2 * tid + 1] += sums[blockIdx.x];

    __syncthreads();

    Y[2 * id] = temp[2 * tid];
    Y[2 * id + 1] = temp[2 * tid + 1];
}

__global__ void exclusive_parallel_scan(float *Y, float *X, float *sums, int unsigned n) {
    extern __shared__ float temp[];

    int tid = threadIdx.x;
    int id = blockDim.x * blockIdx.x + tid;

    temp[2 * tid] = X[2 * id];
    temp[2 * tid + 1] = X[2 * id + 1];

    int offset = 1;
    for (int d = n / 2; d > 0; d >>= 1) {
        __syncthreads();

        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;

            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (tid == 0) {
        if (sums != 0) {
            sums[blockIdx.x] = temp[n - 1];
        }
        temp[n - 1] = 0;
    }

    for (int d = 1; d < n; d <<= 1) {
        offset >>= 1;
        __syncthreads();

        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    Y[2 * id] = temp[2 * tid];
    Y[2 * id + 1] = temp[2 * tid + 1];
}

int main(int argc, char **argv) {
    int N = 2 * BLOCKS * BLOCK_DIM;

    unsigned int bytes = sizeof(float) * N;

    float *h_data = (float *)malloc(bytes);
    for (unsigned int i = 0; i < N; ++i) h_data[i] = 1.0f;

    float *d_in;
    float *d_out;

    float *d_block_sums;

    hipMalloc((void **)&d_in, bytes);
    hipMalloc((void **)&d_out, bytes);

    hipMalloc((void **)&d_block_sums, BLOCKS * sizeof(float));

    hipMemcpy(d_in, h_data, bytes, hipMemcpyHostToDevice);

    exclusive_parallel_scan<<<BLOCKS, BLOCK_DIM, 2 * BLOCK_DIM * sizeof(float)>>>(d_out, d_in, d_block_sums,
                                                                                  2 * BLOCK_DIM);

    exclusive_parallel_scan<<<1, BLOCKS / 2, BLOCKS * sizeof(float)>>>(d_block_sums, d_block_sums, 0, BLOCKS);

    add<<<BLOCKS, BLOCK_DIM, 2 * BLOCK_DIM * sizeof(float)>>>(d_out, d_block_sums, 2 * BLOCK_DIM);

    hipMemcpy(h_data, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf(i == 20 ? "\n" : "");
        printf("%04.0f ", h_data[i]);
    }
    printf("\n");

    free(h_data);

    hipFree(d_out);
    hipFree(d_in);
    hipFree(d_block_sums);

    return 0;
}
