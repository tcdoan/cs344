#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>
#include <algorithm>
#include "CycleTimer.h"

constexpr int N = 2*1024;
__global__ void simple_reduce_kernel(float *d_out, float *d_in)
{
    unsigned int i = threadIdx.x*2;
    for (unsigned int k = 1; k <= blockDim.x; k *= 2) {
        if (threadIdx.x % k == 0) {
            d_in[i] += d_in[i+k];
        } 
        // wait for all adds at one stage are done
        __syncthreads();
    }

    if (i == 0) *d_out = d_in[0];
}

int main(int argc, char **argv)
{
    int IN_BYTES = sizeof(float) *  N;
    int OUT_BYTES = sizeof(float);

    float* h_in= new float[N];
    std::fill_n(h_in, N, 1);

    float *d_in;
    float *d_out;
    
    hipMalloc((void **) &d_in, IN_BYTES);
    hipMalloc((void **) &d_out, OUT_BYTES);

    hipMemcpy(d_in, h_in, IN_BYTES, ::hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();
    simple_reduce_kernel<<<1, N/2>>>(d_out, d_in);
    double endTime = CycleTimer::currentSeconds();    
    printf("Time elapsed %.3f ms \n", 1000.f * (endTime - startTime));

    float h_out;
    hipMemcpy(&h_out, d_out, OUT_BYTES, ::hipMemcpyDeviceToHost);

    printf("sum = %.3f \n", h_out);

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}